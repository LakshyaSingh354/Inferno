#include "hip/hip_runtime.h"
/*
================================================================================
 Fused Kernel: cuBLAS GEMM + ReLU for the Inferno Compiler (Corrected)
================================================================================

This version corrects the `hipblasSgemm` call. The previous error
"parameter number 8 had an illegal value" pointed to an incorrect `lda`
(leading dimension of A).

The fix is to use the standard, robust technique for handling row-major
matrices (like PyTorch's) in column-major libraries (like cuBLAS). We
perform the operation C' = B' * A', where the ' indicates that the
row-major data is being interpreted as column-major. This is equivalent
to the desired C = A * B.

This approach avoids the confusing HIPBLAS_OP_T transpose flags and leads
to a cleaner and more correct implementation.

================================================================================
*/

#include <torch/extension.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <vector>

// Error checking macros
#define CHECK_CUDA(x) AT_ASSERTM(x.ok(), #x " failed: " #x)
#define CHECK_CUBLAS(x) AT_ASSERTM((x) == HIPBLAS_STATUS_SUCCESS, #x " failed!")

// ================================================================================
// SECTION 1: The Custom CUDA ReLU Kernel (Unchanged)
// ================================================================================
__global__ void relu_kernel(float* data, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        data[idx] = fmaxf(data[idx], 0.0f);
    }
}

// ================================================================================
// SECTION 2: The C++ Host Orchestrators
// ================================================================================

// --- CORRECTED Fused GEMM + ReLU Function ---
void fused_gemm_relu_forward_cuda(
    hipblasHandle_t handle,
    torch::Tensor A,
    torch::Tensor B,
    torch::Tensor C) {

    // A is (M, K), B is (K, N), C is (M, N)
    int M = A.size(0);
    int K = A.size(1);
    int N = B.size(1);

    const float alpha = 1.0f;
    const float beta = 0.0f;

    // --- STEP 1: Perform MatMul with cuBLAS (Corrected Call) ---
    // We compute C(M,N) = A(M,K) * B(K,N).
    // In column-major cuBLAS, this is equivalent to C_T(N,M) = B_T(N,K) * A_T(K,M).
    // We pass the matrices in the order (B, A) and tell cuBLAS not to transpose them.
    // cuBLAS will interpret our row-major B(K,N) as a column-major B_T(N,K).
    // cuBLAS will interpret our row-major A(M,K) as a column-major A_T(K,M).
    // The parameters for hipblasSgemm are for the column-major operation:
    // m=N, n=M, k=K
    // lda (for B) is N. ldb (for A) is K. ldc (for C) is N.
    CHECK_CUBLAS(hipblasSgemm(handle,
                             HIPBLAS_OP_N, // Do NOT transpose B
                             HIPBLAS_OP_N, // Do NOT transpose A
                             N, M, K,
                             &alpha,
                             B.data_ptr<float>(), N, // lda for B is N
                             A.data_ptr<float>(), K, // ldb for A is K
                             &beta,
                             C.data_ptr<float>(), N)); // ldc for C is N

    // --- STEP 2: Launch the custom ReLU kernel ---
    const int total_elements = M * N;
    const int threads_per_block = 256;
    const int blocks_per_grid = (total_elements + threads_per_block - 1) / threads_per_block;
    relu_kernel<<<blocks_per_grid, threads_per_block>>>(C.data_ptr<float>(), total_elements);
    
    AT_ASSERTM(hipGetLastError() == hipSuccess, "ReLU kernel launch failed");
}


// --- DEBUG FUNCTION 1: GEMM Only ---
void debug_gemm_only_cuda(
    hipblasHandle_t handle,
    torch::Tensor A,
    torch::Tensor B,
    torch::Tensor C) {

    int M = A.size(0);
    int K = A.size(1);
    int N = B.size(1);
    const float alpha = 1.0f;
    const float beta = 0.0f;

    // This is the same corrected cuBLAS call as above, but without the ReLU.
    CHECK_CUBLAS(hipblasSgemm(handle,
                             HIPBLAS_OP_N, HIPBLAS_OP_N,
                             N, M, K,
                             &alpha,
                             B.data_ptr<float>(), N,
                             A.data_ptr<float>(), K,
                             &beta,
                             C.data_ptr<float>(), N));
}

// --- DEBUG FUNCTION 2: ReLU Only ---
void debug_relu_only_cuda(torch::Tensor T) {
    const int total_elements = T.numel();
    const int threads_per_block = 256;
    const int blocks_per_grid = (total_elements + threads_per_block - 1) / threads_per_block;
    relu_kernel<<<blocks_per_grid, threads_per_block>>>(T.data_ptr<float>(), total_elements);
    AT_ASSERTM(hipGetLastError() == hipSuccess, "ReLU kernel launch failed");
}


// ================================================================================
// SECTION 3: The Pybind11 Wrapper (Updated)
// ================================================================================

hipblasHandle_t get_cublas_handle() {
    static bool initialized = false;
    static hipblasHandle_t handle;
    if (!initialized) {
        CHECK_CUBLAS(hipblasCreate(&handle));
        initialized = true;
    }
    return handle;
}

// Wrapper for the main fused function
torch::Tensor fused_gemm_relu(torch::Tensor A, torch::Tensor B) {
    AT_ASSERTM(A.dim() == 2 && B.dim() == 2, "Inputs must be 2D tensors");
    AT_ASSERTM(A.size(1) == B.size(0), "Matrix dimensions mismatch");
    AT_ASSERTM(A.is_cuda() && B.is_cuda(), "Inputs must be CUDA tensors");

    auto C = torch::empty({A.size(0), B.size(1)}, A.options());
    fused_gemm_relu_forward_cuda(get_cublas_handle(), A, B, C);
    return C;
}

// Wrapper for the GEMM-only debug function
torch::Tensor debug_gemm_only(torch::Tensor A, torch::Tensor B) {
    AT_ASSERTM(A.dim() == 2 && B.dim() == 2, "Inputs must be 2D tensors");
    AT_ASSERTM(A.size(1) == B.size(0), "Matrix dimensions mismatch");
    AT_ASSERTM(A.is_cuda() && B.is_cuda(), "Inputs must be CUDA tensors");

    auto C = torch::empty({A.size(0), B.size(1)}, A.options());
    debug_gemm_only_cuda(get_cublas_handle(), A, B, C);
    return C;
}

// Wrapper for the ReLU-only debug function
torch::Tensor debug_relu_only(torch::Tensor T) {
    AT_ASSERTM(T.is_cuda(), "Input must be a CUDA tensor");
    // We operate in-place for this debug function, so we clone the input
    // to avoid modifying the original tensor passed from Python.
    auto T_out = T.clone();
    debug_relu_only_cuda(T_out);
    return T_out;
}


// --- pybind11 Module Definition (Updated) ---
PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("fused_gemm_relu", &fused_gemm_relu, "Fused GEMM + ReLU forward (CUDA)");
    m.def("debug_gemm_only", &debug_gemm_only, "DEBUG: GEMM only (CUDA)");
    m.def("debug_relu_only", &debug_relu_only, "DEBUG: ReLU only (CUDA)");
}
